#include "hip/hip_runtime.h"
<<<<<<< HEAD
//based on code found at https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/
=======
//based on the code on https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/

>>>>>>> bb60c310d639cf6f0b4edc8b2cfcc10bce86bbb1
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
#include <iostream>

using namespace cv;
using namespace std;

__global__ void red(uint8_t* input, int width, int height, int colorWidthStep)
{
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height))
	{
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		input[color_tid + 0] = 0;
		input[color_tid + 1] = 0;
		
	}
}

inline void mono_red(const Mat& input) {
	const int Bytes = input.step * input.rows;
	uint8_t* d_input;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((uint8_t**)&d_input, sizeof(uint8_t) * Bytes);
	hipMemcpy(d_input, input.data, sizeof(uint8_t) * Bytes, hipMemcpyHostToDevice);
	dim3 block(16, 16);
	dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	hipEventRecord(start, 0);
	red << <grid, block >> > (d_input, input.cols, input.rows, input.step);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(input.data, d_input, sizeof(uint8_t) * Bytes, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();	
}

int main(int argc, char const* argv[]) {

	printf("Program is started\n");
	Mat image = imread("lena.jpg");

	mono_red(image);


	imwrite("lena_red_CUDA.jpg", image);
	system("pause");


	return 0;
}
