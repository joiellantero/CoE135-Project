#include "hip/hip_runtime.h"
//based on the code on https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/

#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
#include <iostream>

using namespace cv;
using namespace std;

__global__ void bgr_to_rgb_kernel(uint8_t* input, int width, int height, int colorWidthStep)
{
	//2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		//const uint8_t t = input[color_tid + 0];
		input[color_tid + 0] = 0;
		input[color_tid + 1] = 0;		
	}
}

inline void mono_red(const Mat& input) {
	const int Bytes = input.step * input.rows;
	uint8_t* d_input;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((uint8_t**)&d_input, sizeof(uint8_t) * Bytes);
	hipMemcpy(d_input, input.data, sizeof(uint8_t) * Bytes, hipMemcpyHostToDevice);
	//hipMemcpy(d_output,output.ptr(),Bytes,hipMemcpyHostToDevice);
	dim3 block(16, 16);
	dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	hipEventRecord(start, 0);
	bgr_to_rgb_kernel << <grid, block >> > (d_input, input.cols, input.rows, input.step);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
//	hipDeviceSynchronize();
	hipMemcpy(input.data, d_input, sizeof(uint8_t) * Bytes, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

int main(int argc, char const* argv[]) {

	printf("Program is started\n");
	Mat image = imread("lena.jpg");
	//Mat image(image_bgr.rows, image_bgr.cols, CV_8UC3);
	//cvtColor(image_bgr, image, COLOR_BGR2RGB);
	//Mat image_out(image.rows, image.cols, CV_8UC3);

	mono_red(image);


	imwrite("lena_red_CUDA.jpg", image);
	system("pause");


	return 0;
}
