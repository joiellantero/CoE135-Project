#include "hip/hip_runtime.h"
<<<<<<< HEAD
//based on code found at https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/
=======
//based on the code on https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/

>>>>>>> bb60c310d639cf6f0b4edc8b2cfcc10bce86bbb1
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
#include <iostream>
#include <time.h>

using namespace cv;
using namespace std;

__global__ void rgb_to_gray_kernel(uint8_t* input, int width, int height, int colorWidthStep)
{
	//2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		int r = input[color_tid + 0];
		int g = input[color_tid + 1];
		int b = input[color_tid + 2];
		input[color_tid + 0] = b * 0.07 + g * 0.71 + r * 0.21;
		input[color_tid + 1] = b * 0.07 + g * 0.71 + r * 0.21;
		input[color_tid + 2] = b * 0.07 + g * 0.71 + r * 0.21;
	}
}

inline void rgb_to_gray(const Mat& input) {
	const int Bytes = input.step * input.rows;
	uint8_t* d_input;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((uint8_t**)&d_input, sizeof(uint8_t) * Bytes);
	hipMemcpy(d_input, input.data, sizeof(uint8_t) * Bytes, hipMemcpyHostToDevice);
	dim3 block(4, 4);
	dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	hipEventRecord(start, 0);
	rgb_to_gray_kernel << <grid, block >> > (d_input, input.cols, input.rows, input.step);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(input.data, d_input, sizeof(uint8_t) * Bytes, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
}

int main(int argc, char const* argv[]) {

	printf("Program is started\n");
	Mat image = imread("lena.jpg");
	
	rgb_to_gray(image);

	imwrite("lena_grayscale_CUDA.jpg", image);
	system("pause");

	return 0;
}
