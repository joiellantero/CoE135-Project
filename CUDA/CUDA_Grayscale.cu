#include "hip/hip_runtime.h"
//based on code found at https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <string.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
#include <dirent.h>
#include <iostream>
#include <cstring>
#include <sys/types.h>
#include <stdlib.h>

__global__ void rgb_to_gray_kernel(uint8_t* input, int width, int height, int colorWidthStep);

using namespace cv;
using namespace std;

__global__ void rgb_to_gray_kernel(uint8_t* input, int width, int height, int colorWidthStep)
{
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	if ((xIndex < width) && (yIndex < height))
	{
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		int r = input[color_tid + 0];
		int g = input[color_tid + 1];
		int b = input[color_tid + 2];
		input[color_tid + 0] = b * 0.07 + g * 0.71 + r * 0.21;
		input[color_tid + 1] = b * 0.07 + g * 0.71 + r * 0.21;
		input[color_tid + 2] = b * 0.07 + g * 0.71 + r * 0.21;
	}
}

float rgb_to_gray(const Mat& input) {
	const int Bytes = input.step * input.rows;
	uint8_t* d_input;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((uint8_t**)&d_input, sizeof(uint8_t) * Bytes);
	hipMemcpy(d_input, input.data, sizeof(uint8_t) * Bytes, hipMemcpyHostToDevice);
	dim3 block(4, 4);
	dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	hipEventRecord(start, 0);
	rgb_to_gray_kernel << <grid, block>> > (d_input, input.cols, input.rows, input.step);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(input.data, d_input, sizeof(uint8_t) * Bytes, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
	return time;
}

int main()
{
	struct dirent* de;  // Pointer for directory entry 
	int i = 0;
	float time = 0;
	FILE* fp;

	// opendir() returns a pointer of DIR type.  
	DIR* dr = opendir("./test_images/");
	printf("Program has started\n");
	if (dr == NULL)  // opendir returns NULL if couldn't open directory 
	{
		printf("Could not open current directory");
		return 0;

	}
	while ((de = readdir(dr)) != NULL) {
		if (i > 1) {
			string name(de->d_name);
			string path("./test_images/");
			string new_path("./processed_images/");
			path.append(name);
			new_path.append(name);
			cout << path << endl;
			Mat image = imread(path);

			time = rgb_to_gray(image);

			imwrite(new_path, image);
			fp = fopen("CUDA_grayscale.txt", "a");
			fprintf(fp, "%f\n", time / 1000);
			fclose(fp);
		}
		i++;
	}
	closedir(dr);
	printf("End.");
	system("pause");
}
