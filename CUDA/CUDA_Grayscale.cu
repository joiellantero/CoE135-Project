#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
#include <iostream>
#include <time.h>

using namespace cv;
using namespace std;

__global__ void rgb_to_gray_kernel(uint8_t* input, int width, int height, int colorWidthStep)
{
	//2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		int r = input[color_tid + 0];
		int g = input[color_tid + 1];
		int b = input[color_tid + 2];
		input[color_tid + 0] = b * 0.07 + g * 0.71 + r * 0.21;
		input[color_tid + 1] = b * 0.07 + g * 0.71 + r * 0.21;
		input[color_tid + 2] = b * 0.07 + g * 0.71 + r * 0.21;
	}
}

inline void rgb_to_gray(const Mat& input) {
	const int Bytes = input.step * input.rows;
	uint8_t* d_input;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((uint8_t**)&d_input, sizeof(uint8_t) * Bytes);
	hipMemcpy(d_input, input.data, sizeof(uint8_t) * Bytes, hipMemcpyHostToDevice);
	//hipMemcpy(d_output,output.ptr(),Bytes,hipMemcpyHostToDevice);
	dim3 block(16, 16);
	//dim3 threads(4,1,1);
	dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	//dim3 grid((input.cols / block.x)+1, (input.rows / block.y)+1);
	//dim3 threadsPerBlock(4, 4,1);
	//dim3 numBlocks(ceil((float)input.cols / threadsPerBlock.x), ceil((float)input.rows / threadsPerBlock.y),1);
	hipEventRecord(start, 0);
	rgb_to_gray_kernel << <grid, block >> > (d_input, input.cols, input.rows, input.step);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(input.data, d_input, sizeof(uint8_t) * Bytes, hipMemcpyDeviceToHost);
	hipFree(d_input);
	//	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
}

int main(int argc, char const* argv[]) {

	printf("Program is started\n");
	Mat image = imread("lena.jpg");
	//Mat image(image_bgr.rows, image_bgr.cols, CV_8UC3);
	//cvtColor(image_bgr, image, COLOR_BGR2RGB);
	//Mat image_out(image.rows, image.cols, CV_8UC3);
	rgb_to_gray(image);

	imwrite("lena_grayscale_CUDA_2.jpg", image);
	system("pause");

	return 0;
}