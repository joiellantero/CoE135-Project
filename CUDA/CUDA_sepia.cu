#include "hip/hip_runtime.h"
//based on the code on https://www.ylmzcmlttn.com/2019/06/07/bgr-to-rgb-with-cuda-cuda-and-opencv/

#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include <opencv2\core.hpp>
#include <opencv2\highgui.hpp>
#include <opencv2\imgproc.hpp>
#include <iostream>

using namespace cv;
using namespace std;

__global__ void sepia(uint8_t* input, int width, int height, int colorWidthStep)
{
	//2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	//Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
		//const uint8_t t = input[color_tid + 0];
		int b = input[color_tid + 0];
		int g = input[color_tid + 1];
		int r = input[color_tid + 2];
		int tr = (int)(r * 0.393 + g * 0.769 + b * 0.189);
		int tg = (int)(r * 0.349 + g * 0.686 + b * 0.168);
		int tb = (int)(r * 0.272 + g * 0.534 + b * 0.131);
		if (tr > 255) {
			r = 255;
		}else {
			r = tr;
		}
		if (tg > 255) {
			g = 255;
		}
		else {
			g = tg;
		}
		if (tb > 255) {
			b = 255;
		}
		else {
			b = tb;
		}
		input[color_tid + 0] = b;
		input[color_tid + 1] = g;
		input[color_tid + 2] = r;
	}
}

inline void mono_red(const Mat& input) {
	const int Bytes = input.step * input.rows;
	uint8_t* d_input;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipMalloc((uint8_t**)&d_input, sizeof(uint8_t) * Bytes);
	hipMemcpy(d_input, input.data, sizeof(uint8_t) * Bytes, hipMemcpyHostToDevice);
	//hipMemcpy(d_output,output.ptr(),Bytes,hipMemcpyHostToDevice);
	dim3 block(4, 4);
	dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	hipEventRecord(start, 0);
	sepia << <grid, block >> > (d_input, input.cols, input.rows, input.step);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipMemcpy(input.data, d_input, sizeof(uint8_t) * Bytes, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipEventElapsedTime(&time, start, stop);
	printf("Time for the kernel: %f ms\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//hipDeviceSynchronize();
}

int main(int argc, char const* argv[]) {

	printf("Program is started\n");
	Mat image = imread("lena.jpg");
	//Mat image(image_bgr.rows, image_bgr.cols, CV_8UC3);
	//cvtColor(image_bgr, image, COLOR_BGR2RGB);
	//Mat image_out(image.rows, image.cols, CV_8UC3);

	mono_red(image);


	imwrite("lena_sepia_CUDA.jpg", image);
	system("pause");


	return 0;
}
